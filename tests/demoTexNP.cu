#include "hip/hip_runtime.h"
/**
demoTexNP 
==========

Adapted from the below adding use of NP.hh rather than thrust 

* ~/intro_to_cuda/textures/demoTex.cu 
* https://bitbucket.org/simoncblyth/intro_to_cuda/src/master/textures/demoTex.cu

CUDA Docs
--------------

* http://on-demand.gputechconf.com/gtc-express/2011/presentations/texture_webinar_aug_2011.pdf
* https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#texture-fetching
* https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#table-lookup


Textures, Samples and the 0.5f offset
-----------------------------------------

Consider a 4x4 texture. The array elements would be addressed with integers::

   ix: 0,1,2,3 
   iy: 0,1,2,3 

BUT: in order to retrieve sample values, need to offset float texture coordinates by 0.5::

   x:0.5,1.5,2.5,3.5 
   y:0.5,1.5,2.5,3.5 

So to work with the texture need to realize that the sample values are mid-texel ones and arrange 
the contents of the array (the samples) accordingly.::

      +--------+--------+--------+--------+
      | .5 .5  | 1.5 .5 | 2.5 .5 | 3.5 .5 |
      |   +    |   +    |   +    |   +    | 
      |        |        |        |        |
      +--------+--------+--------+--------+
      | .5 1.5 | 1.5 1.5| 2.5 1.5| 3.5 1.5|
      |   +    |   +    |   +    |   +    |  
      |        |        |        |        |
      +--------+--------+--------+--------+
      | .5 2.5 | 1.5 2.5| 2.5 2.5| 3.5 2.5|
      |   +    |   +    |   +    |   +    |  
      |        |        |        |        |
      +--------+--------+--------+--------+
      | .5 3.5 | 1.5 3.5| 2.5 3.5| 3.5 3.5|
      |   +    |   +    |   +    |   +    |  
      |        |        |        |        |
      +--------+--------+--------+--------+


Top left origin is considered as that more naturally 
to the array serialization, with the first element 
in the array corresponding to texel at top left, 
and the last corresponding to the element at bottom right. 

Thus when creating arrays it is more natural to order 
dimensions (ni,nj) = (height, width) with height before width::

   NP* a = NP::Make<float>( height, width ) 

texture docs
---------------

https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#texture-memory

Whether texture coordinates are normalized or not. By default, textures are
referenced (by the functions of Texture Functions) using floating-point
coordinates in the range [0, N-1] where N is the size of the texture in the
dimension corresponding to the coordinate. For example, a texture that is 64x32
in size will be referenced with coordinates in the range [0, 63] and [0, 31]
for the x and y dimensions, respectively. Normalized texture coordinates cause
the coordinates to be specified in the range [0.0, 1.0-1/N] instead of [0,
N-1], so the same 64x32 texture would be addressed by normalized coordinates in
the range [0, 1-1/N] in both the x and y dimensions. Normalized texture
coordinates are a natural fit to some applications' requirements, if it is
preferable for the texture coordinates to be independent of the texture size.

The addressing mode. It is valid to call the device functions of Section B.8
with coordinates that are out of range. The addressing mode defines what
happens in that case. The default addressing mode is to clamp the coordinates
to the valid range: [0, N) for non-normalized coordinates and [0.0, 1.0) for
normalized coordinates. If the border mode is specified instead, texture
fetches with out-of-range texture coordinates return zero. For normalized
coordinates, the wrap mode and the mirror mode are also available. When using
the wrap mode, each coordinate x is converted to frac(x)=x - floor(x) where
floor(x) is the largest integer not greater than x. When using the mirror mode,
each coordinate x is converted to frac(x) if floor(x) is even and 1-frac(x) if
floor(x) is odd. The addressing mode is specified as an array of size three
whose first, second, and third elements specify the addressing mode for the
first, second, and third texture coordinates, respectively; the addressing mode
are hipAddressModeBorder, hipAddressModeClamp, hipAddressModeWrap, and
hipAddressModeMirror; hipAddressModeWrap and hipAddressModeMirror are only
supported for normalized texture coordinates


**/

#include "NP.hh"

__global__ void demoTexNP(float* output, hipTextureObject_t texObj, unsigned width, unsigned height)
{   
    unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned iy = blockIdx.y * blockDim.y + threadIdx.y;

    if( ix >= width || iy >= height ) return ; 

    // adding 0.5 avoids interpolation giving directly the mid-texel sample value
    float x = float(ix) + 0.5f ;
    float y = float(iy) + 0.5f ;

    //float v = x ; 
    //float v = y ; 
    float v = tex2D<float>(texObj, x, y ); 

    output[iy * width + ix] = v  ; 

    printf("//demoTexNP : Thread index: (ix,iy) (%i, %i) (x,y) (%10.4f, %10.4f)    v = %f\n", ix, iy, x, y, v ); 
}


/**
uploadTexture
---------------

* ~/opticks/sysrap/tests/SIMGStandaloneTest.cu
* https://docs.nvidia.com/cuda/cuda-runtime-api/structcudaTextureDesc.html

Wrap
Mirror
    hipAddressModeWrap and hipAddressModeMirror are only supported 
    for normalized texture coordinates

Clamp

Border

**/

hipTextureObject_t uploadTexture( const NP* a )
{
    unsigned height = a->shape[0] ; 
    unsigned width = a->shape[1] ; 

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, width, height);
    hipMemcpyToArray(cuArray, 0, 0, a->bytes(), a->arr_bytes(), hipMemcpyHostToDevice);
    /// ^^^^^^^^^^^ DEPRECATED METHOD    
 
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray; 

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));

    hipTextureAddressMode x_addressMode = hipAddressModeClamp ;  
    hipTextureAddressMode y_addressMode = hipAddressModeClamp ;  

    texDesc.addressMode[0] = x_addressMode ;
    texDesc.addressMode[1] = y_addressMode ;

    hipTextureFilterMode filterMode = hipFilterModeLinear ;  
    //hipTextureFilterMode filterMode = hipFilterModePoint ;    // do not interpolate 
    texDesc.filterMode = filterMode ; 

    // specifies whether integer data should be converted to floating point or not.
    // Note that this applies only to 8-bit and 16-bit integer formats. 
    // 32-bit integer format would not be promoted, regardless of whether or not this is set 
    //hipTextureReadMode readMode = hipReadModeNormalizedFloat ; 
    hipTextureReadMode readMode = hipReadModeElementType ;  // return data of the type of the underlying buffer

    texDesc.readMode = readMode ;  

    //texDesc.normalizedCoords = 1 ;            // addressing into the texture with floats in range 0:1
    texDesc.normalizedCoords = 0 ;         
    
    hipTextureObject_t texObj = 0; 
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

    return texObj ; 
}

int main()
{
    NP* a = NP::Make<float>(4, 4); 
    a->fillIndexFlat();  

    unsigned height = a->shape[0] ; 
    unsigned width = a->shape[1] ; 

    hipTextureObject_t texObj = uploadTexture(a); 

    NP* b = NP::Make<float>(4, 4);  
    // not necessarily the same dimension as *a* (that is the point of interpolation)

    float* d_output;
    hipMalloc(&d_output, b->arr_bytes() );

    dim3 block(16,16); 
    dim3 grid((width + block.x - 1)/block.x, (height + block.y - 1)/block.y);
    printf(" block (%d, %d) grid (%d, %d) \n", block.x, block.y, grid.x, grid.y ); 

    demoTexNP<<<grid, block>>>(d_output, texObj, width, height);    
    hipMemcpy(b->bytes(), d_output, b->arr_bytes(),  hipMemcpyDeviceToHost);

    const char* FOLD = "/tmp/demoTexNP" ; 
    hipDeviceSynchronize();  // without Synchronize the process terminates before printf output appears 

    a->save(FOLD, "a.npy"); 
    b->save(FOLD, "b.npy"); 

    return 0 ; 
}

